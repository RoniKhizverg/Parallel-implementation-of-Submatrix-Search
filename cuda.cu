#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "hip/hip_runtime.h"


__device__ double calcMatchSum(int* picture, int* object, int i,int j,double matchingValue, int pictureSize, int objectSize) {
	double absSum = 0;
	int column = j;
	for (int k = 0; k < objectSize; k++, i++) {
		column = j;
		for (int x = 0; x < objectSize; x++, column++) {
		
			absSum = absSum + fabs((double)((picture[i * pictureSize + column] - object[k * objectSize + x]) / (double)picture[i * pictureSize + column]));
			
			/*** If we will use thr condition below, the result will be better, but the function will not consider as an "heavy function". ***/
			
			//if(absSum>matchingValue){
			//	return absSum;
			//}
		}
	}
	return absSum;
}

__global__ void calcAbsSum(int pictureSize,int objectSize,int* picture,int* object, double* absArray, int threadsPerBlock,double matchingValue) {
	double matcing = 0;

	for(int i = threadIdx.x; i < pictureSize; i += threadsPerBlock) // after 32 threads, we will return to the first thread again
	{
		for(int j = threadIdx.y; j < pictureSize; j += threadsPerBlock) // after 32 threads, we will return to the first thread again
		{
			matcing = calcMatchSum(picture, object, i, j ,matchingValue, pictureSize, objectSize);
			absArray[i*pictureSize + j] = matcing;
		}
	}
}

void findPairOnGPU( Element* picture,  Element** object , double matchingValue, Pair* pair, int* Count, int numOfObjects)
{
	pair->i =-1;
	pair->j = -1;
	pair->pictureID = picture->Id;
	pair->objectID = -1;
	pair-> isPair = 0;

	int foundObj = 0;
	
   	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
  
    	int *d_Picture;

   	// Allocate memory on GPU to copy the data from the host
    	err = hipMalloc((void **)&d_Picture, (picture->dimention * picture->dimention) * sizeof(int));
    	if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
    	    	    	
    	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_Picture, picture->members, (picture->dimention * picture->dimention) * sizeof(int), hipMemcpyHostToDevice);
    	if (err != hipSuccess) {  
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
    	
    	for(int obj = 0; obj < numOfObjects; obj++) {
    		int *d_Object;
    		
	    	double *d_absArray;
	    	double *h_absArray;
	    	
	    	h_absArray = (double*)malloc((picture->dimention*picture->dimention)* sizeof(double));
	    	if (h_absArray == NULL) {
				printf("Problem to allocate memory\n");
				exit(0);
			}
	    	
			//Allocate memory on GPU to copy the data from the host
    		err = hipMalloc((void **)&d_Object, object[obj]->dimention * object[obj]->dimention * sizeof(int));
    		if (err != hipSuccess){
				fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
		
    		// Allocate memory on GPU to copy the data from the host
    		err = hipMalloc((void **)&d_absArray, (picture->dimention*picture->dimention)* sizeof(double));
	    	if (err != hipSuccess){
				fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

    		// Allocate memory on GPU to copy the data from the host
	    	err = hipMemcpy(d_Object, object[obj]->members, (object[obj]->dimention * object[obj]->dimention) * sizeof(int), hipMemcpyHostToDevice);
	    	if (err != hipSuccess){
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
			}

	    	// Launch the Kernel
	    	int threadsPerBlock = 32;
	    	if(picture->dimention <= threadsPerBlock)
	    	{
	    		dim3 dimBlock(picture->dimention, picture->dimention);
		    	calcAbsSum<<<1, dimBlock>>>(picture->dimention, object[obj]->dimention, d_Picture,d_Object, d_absArray,threadsPerBlock,matchingValue);
		    	err = hipGetLastError();
				if (err != hipSuccess){
					fprintf(stderr, "Failed to launch calcSum kernel (error code %s)!\n", hipGetErrorString(err));
					exit(EXIT_FAILURE);
				}
			}
			else
			{
					dim3 dimBlock(threadsPerBlock, threadsPerBlock);
					calcAbsSum<<<1, dimBlock>>>(picture->dimention, object[obj]->dimention, d_Picture, d_Object, d_absArray,threadsPerBlock,matchingValue);
					err = hipGetLastError();
				if (err != hipSuccess){
					fprintf(stderr, "Failed to launch calcSum kernel (error code %s)!\n", hipGetErrorString(err));
					exit(EXIT_FAILURE);
				}
			}
	    	
	    	
    		err = hipMemcpy(h_absArray, d_absArray, (picture->dimention*picture->dimention) * sizeof(double), hipMemcpyDeviceToHost);
			if (err != hipSuccess){
				fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
		 
			for(int i = 0; i < picture->dimention && !foundObj; i++ ){
		
				for(int j = 0; j <picture->dimention && !foundObj; j++){
			
					if(h_absArray[i*picture->dimention + j] <= matchingValue){
					 
						pair->i = i;
						pair->j = j;
						pair->pictureID = picture->Id;
						pair->objectID = object[obj]->Id;
						pair-> isPair = 1;
						foundObj=1;										
					}
				}
			}
	    	
	        err = hipFree(d_Object);
			if (err != hipSuccess){
				fprintf(stderr, "Failed to free device d_Object (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
	        
	        err = hipFree(d_absArray);
	        if (err != hipSuccess){
				fprintf(stderr, "Failed to free device d_SumArray (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
	        
	        free(h_absArray);
	        
	        if(foundObj)
	        	break;
	}
	*Count+=1;	 
	
    // Free allocated memory on GPU
    err = hipFree(d_Picture);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to free device d_SumArray (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
}

